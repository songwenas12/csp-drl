#include "hip/hip_runtime.h"
#include "nn/tanh.h"
#include "tensor/gpu_handle.h"
#include "tensor/gpu_unary_functor.h"

namespace gnn
{

template<typename Dtype>
void TanhAct(DTensor<GPU, Dtype>& in, DTensor<GPU, Dtype>& out)
{
	out.CopyFrom(in);
	UnaryEngine<GPU>::Exec<UnaryTanh>(out.data->ptr, out.shape.Count());
}

template void TanhAct(DTensor<GPU, float>& in, DTensor<GPU, float>& out);
template void TanhAct(DTensor<GPU, double>& in, DTensor<GPU, double>& out);

template<typename Dtype>
__global__ void TanhDerivKernel(Dtype *dst, Dtype *out, Dtype* cur_grad, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        dst[i] += cur_grad[i] * (1 - out[i] * out[i]);
    }
}

template<typename Dtype>
void TanhDeriv(DTensor<GPU, Dtype>& dst, DTensor<GPU, Dtype>& cur_output, DTensor<GPU, Dtype>& cur_grad)
{
	int thread_num = c_uCudaThreadNum;
	if (dst.shape.Count() < thread_num)
		thread_num = dst.shape.Count();
    int blocksPerGrid = (dst.shape.Count() + thread_num - 1) / thread_num;
    TanhDerivKernel <<< blocksPerGrid, thread_num, 0, hipStreamPerThread >>>(dst.data->ptr, cur_output.data->ptr, cur_grad.data->ptr, dst.shape.Count());
}

template void TanhDeriv(DTensor<GPU, float>& dst, DTensor<GPU, float>& cur_output, DTensor<GPU, float>& cur_grad);
template void TanhDeriv(DTensor<GPU, double>& dst, DTensor<GPU, double>& cur_output, DTensor<GPU, double>& cur_grad);


}