#include "hip/hip_runtime.h"
#include "nn/in_top_k.h"
#include "tensor/gpu_handle.h"
#include "tensor/gpu_reduce_kernel.h"
#include "tensor/cuda_helper.h"

namespace gnn
{

template<typename Dtype>
__global__ void InTopkKernel(int* dst, Dtype* pred_prob, int* label_ptr, int cols, int k)
{
    __shared__ int buffer[REDUCE_THREADS];

    Dtype* prob_row = pred_prob + blockIdx.x * cols;

    int i_start = threadIdx.x;
    int i_end = cols;
    int i_step = blockDim.x;    
    buffer[threadIdx.x] = 0;
    int target = label_ptr[blockIdx.x];
    for (int i = i_start; i < i_end; i += i_step)
    {
    	if (i != target && prob_row[i] > prob_row[target])
    		buffer[threadIdx.x]++;
    }
    __syncthreads();

    int shift;
    for (int i = REDUCE_THREAD_BITS - 1; i >= 0; --i)
    {
    	shift = 1 << i;
    	if (threadIdx.x < shift && threadIdx.x + shift < cols)
    	{
    		buffer[threadIdx.x] += buffer[threadIdx.x + shift];
    	}
		__syncthreads();
    }
    if (threadIdx.x == 0)
    	dst[blockIdx.x] = buffer[0] < k;
}

template<typename Dtype>
void IsInTopK(DTensor<GPU, Dtype>& pred, DTensor<GPU, int>& label, DTensor<GPU, int>& out, int k)
{
	assert(pred.rank() == 2, "predicted prob(or logits) should be a matrix");
	assert(pred.rows() == label.shape.Count(), "# instances doesn't match");
	out.Reshape(label.shape.dims);
	dim3 blocks(pred.rows());
	dim3 threads(REDUCE_THREADS);
    InTopkKernel<<<blocks, threads, 0, hipStreamPerThread>>> (out.data->ptr, pred.data->ptr, label.data->ptr, pred.cols(), k);
}

template void IsInTopK(DTensor<GPU, float>& pred, DTensor<GPU, int>& label, DTensor<GPU, int>& out, int k);
template void IsInTopK(DTensor<GPU, double>& pred, DTensor<GPU, int>& label, DTensor<GPU, int>& out, int k);

}