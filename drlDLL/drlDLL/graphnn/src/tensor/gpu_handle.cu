#include "hip/hip_runtime.h"
#include "tensor/gpu_handle.h"
#include "util/gnn_macros.h"
#include "util/mem_holder.h"
#include "tbb/tbb.h"
#include <assert.h>

namespace gnn
{

__global__ void SetupRandKernel(hiprandState_t *state, unsigned long long seed) 
{
    const unsigned int tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    /* Each thread gets same seed, a different sequence number,
     no offset */
    hiprand_init(seed, tidx, 0, &state[tidx]);
}

void GpuHandle::Init(int dev_id, unsigned int _streamcnt)
{
	tbb::task_scheduler_init init(4);
	streamcnt = _streamcnt;
	hipDeviceReset();
	hipSetDevice(dev_id);

	cublashandles = new hipblasHandle_t[streamcnt];
	cusparsehandles = new hipsparseHandle_t[streamcnt];
	inUse = new bool[streamcnt];
	while (!resources.empty())
		resources.pop();
	for (unsigned int id = 0; id < streamcnt; ++id)
	{
		hipblasCreate(&cublashandles[id]);	
		hipsparseCreate(&cusparsehandles[id]);
		inUse[id] = false;
		resources.push(id);
	}
	hipStreamCreate(&cudaRandStream);

	hiprandCreateGenerator(&curandgenerator, HIPRAND_RNG_PSEUDO_DEFAULT);
	
	hiprandSetPseudoRandomGeneratorSeed(curandgenerator, time(NULL));
	
    hipMalloc((void **)&devRandStates, NUM_RND_STREAMS * sizeof(hiprandState_t));
	SetupRandKernel<<<NUM_RND_BLOCKS, NUM_RND_THREADS_PER_BLOCK>>>(devRandStates, 1 + time(NULL)*2);
}

GpuContext GpuHandle::AquireCtx()
{
	r_loc.lock();
	assert(resources.size(), "running out of gpu resources");

	int cur_pos = resources.front();
	resources.pop();

	r_loc.unlock();

	assert(!inUse[cur_pos], "logic error: in-use resource is found available");
	inUse[cur_pos] = true;	
	hipblasSetStream(cublashandles[cur_pos], hipStreamPerThread);
	hipsparseSetStream(cusparsehandles[cur_pos], hipStreamPerThread);
	return GpuContext(cur_pos, cublashandles[cur_pos], cusparsehandles[cur_pos]);
}

void GpuHandle::ReleaseCtx(const GpuContext& ctx)
{
	r_loc.lock();
	resources.push(ctx.id);
	assert(inUse[ctx.id], "logic error: in-use resource is not recorded, or you are releasing same resource multiple times");
	inUse[ctx.id] = false;
	r_loc.unlock();	
}

void GpuHandle::Destroy()
{
	hipDeviceSynchronize();
	hipStreamDestroy(cudaRandStream);
	for (unsigned int id = 0; id < streamcnt; ++id)
	{
		hipblasDestroy(cublashandles[id]);
		hipsparseDestroy(cusparsehandles[id]);
	}
	delete[] cublashandles;
	delete[] cusparsehandles;
	delete[] inUse;
	hiprandDestroyGenerator(curandgenerator);
    hipFree(devRandStates);
	streamcnt = 0U;
	std::cout << "GPU Handle destroyed." << std::endl;
}

hiprandState_t* GpuHandle::devRandStates = NULL;
hipblasHandle_t* GpuHandle::cublashandles = NULL;
hipsparseHandle_t* GpuHandle::cusparsehandles = NULL;
hiprandGenerator_t GpuHandle::curandgenerator;
unsigned int GpuHandle::streamcnt = 1U;
std::queue< int > GpuHandle::resources;
std::mutex GpuHandle::r_loc;
std::mutex GpuHandle::rand_lock;
bool* GpuHandle::inUse = NULL;
hipStream_t GpuHandle::cudaRandStream;

}